#include "hip/hip_runtime.h"
#include "disparity_method.h"

static hipStream_t stream1, stream2, stream3;//, stream4, stream5, stream6, stream7, stream8;
static uint8_t *d_im0;
static uint8_t *d_im1;
static cost_t *d_transform0;
static cost_t *d_transform1;
static uint8_t *d_cost;
static uint8_t *d_disparity;
static uint8_t *d_disparity_filtered_uchar;
static uint8_t *h_disparity;
static uint8_t *d_mmcost;
static uint16_t *d_S;
static uint8_t *d_L0;
static uint8_t *d_L1;
static uint8_t *d_L2;
static uint8_t *d_L3;
static uint32_t cols, rows, size, size_cube_l,size_ppparams;
static uint8_t *pparamsgpu; 

void disparity_errors(cv::Mat estimation, cv::Mat gt_disp, int *n, int *n_err)
{
	int nlocal = 0;
	int nerrlocal = 0;
	
	if(!gt_disp.data) {
		std::cerr << "Couldn't read the gt_disp file " << std::endl;
		exit(EXIT_FAILURE);
	}
	if(estimation.rows != gt_disp.rows || estimation.cols != gt_disp.cols) {
		std::cerr << "Ground truth must have the same dimesions" << std::endl;
		exit(EXIT_FAILURE);
	}
	const int type = estimation.type();
	const uchar depth = type & CV_MAT_DEPTH_MASK;
	for(int i = 0; i < gt_disp.rows; i++) {
		for(int j = 0; j < gt_disp.cols; j++) {
			const uint16_t gt = gt_disp.at<uint16_t>(i, j);
			if(gt > 0) {
				const float gt_f = ((float)gt)/256.0f;
				float est;
				if(depth == CV_8U) {
					est = (float) estimation.at<uint8_t>(i, j);
				} else {
					est = estimation.at<float>(i, j);
				}
				const float err = fabsf(est-gt_f);
				const float ratio = err/fabsf(gt_f);
				if(err > ABS_THRESH && ratio > REL_THRESH) {
					nerrlocal++;
				}
				nlocal++;
			}
		}
	}
	*n += nlocal;
	*n_err += nerrlocal;
}

void init_disparity_method(bool &first_alloc)
{
	// Create streams
	CUDA_CHECK_RETURN(hipStreamCreate(&stream1));
	CUDA_CHECK_RETURN(hipStreamCreate(&stream2));
	CUDA_CHECK_RETURN(hipStreamCreate(&stream3));
	first_alloc = true;
    rows = 0;
    cols = 0;
}

cv::Mat compute_disparity_method(cv::Mat left, cv::Mat right, uint8_t *&d_mcost,uint8_t *pparams,bool &first_alloc)
{
	if(cols != left.cols || rows != left.rows)
	 {
		if(!first_alloc) 
		{
			free_memory();
		}
		first_alloc = false;
		cols = left.cols;
		rows = left.rows;
		size = rows*cols;
		size_cube_l = size*MAX_DISPARITY;    ///// MAX_DISPARITY =128
		size_ppparams = size*8;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_transform0, sizeof(cost_t)*size));

		CUDA_CHECK_RETURN(hipMalloc((void **)&d_transform1, sizeof(cost_t)*size));

		int size_cube = size*MAX_DISPARITY;
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_cost, sizeof(uint8_t)*size_cube));
        d_mcost = d_cost;
        
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_im0, sizeof(uint8_t)*size));

		CUDA_CHECK_RETURN(hipMalloc((void **)&d_im1, sizeof(uint8_t)*size));

		CUDA_CHECK_RETURN(hipMalloc((void **)&d_S, sizeof(uint16_t)*size_cube_l));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_L0, sizeof(uint8_t)*size_cube_l));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_L1, sizeof(uint8_t)*size_cube_l));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_L2, sizeof(uint8_t)*size_cube_l));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_L3, sizeof(uint8_t)*size_cube_l));

		CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity, sizeof(uint8_t)*size));
		CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity_filtered_uchar, sizeof(uint8_t)*size));
		
		CUDA_CHECK_RETURN(hipMalloc((void **)&pparamsgpu, sizeof(uint8_t)*size_ppparams));
		h_disparity = new uint8_t[size];
		d_mmcost = new uint8_t[size_cube_l];
	}
	
	CUDA_CHECK_RETURN(hipMemcpyAsync(pparamsgpu, pparams, sizeof(uint8_t)*size_ppparams, hipMemcpyHostToDevice, stream1));
	CUDA_CHECK_RETURN(hipMemcpyAsync(d_im0, left.ptr<uint8_t>(), sizeof(uint8_t)*size, hipMemcpyHostToDevice, stream1));
	CUDA_CHECK_RETURN(hipMemcpyAsync(d_im1, right.ptr<uint8_t>(), sizeof(uint8_t)*size, hipMemcpyHostToDevice, stream1));

	dim3 block_size;
	block_size.x = 32;
	block_size.y = 32;

	dim3 grid_size;
	grid_size.x = (cols+block_size.x-1) / block_size.x;
	grid_size.y = (rows+block_size.y-1) / block_size.y;

	CenterSymmetricCensusKernelSM2<<<grid_size, block_size, 0, stream1>>>(d_im0, d_im1, d_transform0, d_transform1, rows, cols);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
	{
		printf("Error: %s %d\n", hipGetErrorString(err), err);
		exit(-1);
	}

	// Hamming distance
	CUDA_CHECK_RETURN(hipStreamSynchronize(stream1));
	HammingDistanceCostKernel<<<rows, MAX_DISPARITY, 0, stream1>>>(d_transform0, d_transform1, d_cost, rows, cols);
	err = hipGetLastError();
	if (err != hipSuccess) 
	{
		printf("Error: %s %d\n", hipGetErrorString(err), err);
		exit(-1);
	}

	// Cost Aggregation
	const int PIXELS_PER_BLOCK = COSTAGG_BLOCKSIZE/WARP_SIZE; ////   64/32
	const int PIXELS_PER_BLOCK_HORIZ = COSTAGG_BLOCKSIZE_HORIZ/WARP_SIZE;

	CostAggregationKernelLeftToRight<<<(rows+PIXELS_PER_BLOCK_HORIZ-1)/PIXELS_PER_BLOCK_HORIZ, COSTAGG_BLOCKSIZE_HORIZ, 0, stream2>>>(d_cost, d_L0, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, pparamsgpu);
	err = hipGetLastError();
	if (err != hipSuccess) 
	{
		printf("Error: %s %d\n", hipGetErrorString(err), err);
		exit(-1);
	}

	CostAggregationKernelRightToLeft<<<(rows+PIXELS_PER_BLOCK_HORIZ-1)/PIXELS_PER_BLOCK_HORIZ, COSTAGG_BLOCKSIZE_HORIZ, 0, stream3>>>(d_cost, d_L1,rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, pparamsgpu);
	err = hipGetLastError();
	if (err != hipSuccess) 
	{
		printf("Error: %s %d\n", hipGetErrorString(err), err);
		exit(-1);
	}

	CostAggregationKernelUpToDown<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L2, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, pparamsgpu);
	err = hipGetLastError();
	if (err != hipSuccess) 
	{
		printf("Error: %s %d\n", hipGetErrorString(err), err);
		exit(-1);
	}
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	CostAggregationKernelDownToUp<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, stream1>>>(d_cost, d_L3, rows, cols, d_transform0, d_transform1, d_disparity, d_L0, d_L1, d_L2, d_L3, pparamsgpu);
	err = hipGetLastError();
	if (err != hipSuccess) 
	{
		printf("Error: %s %d\n", hipGetErrorString(err), err);
		exit(-1);
	}

	MedianFilter3x3<<<(size+MAX_DISPARITY-1)/MAX_DISPARITY, MAX_DISPARITY, 0, stream1>>>(d_disparity, d_disparity_filtered_uchar, rows, cols);
	err = hipGetLastError();
	if (err != hipSuccess) 
	{
		printf("Error: %s %d\n", hipGetErrorString(err), err);
		exit(-1);
	}

	CUDA_CHECK_RETURN(hipMemcpy(h_disparity, d_disparity_filtered_uchar, sizeof(uint8_t)*size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(d_mmcost,d_cost, sizeof(uint8_t)*size_cube_l, hipMemcpyDeviceToHost));
    d_mcost = d_mmcost;
	cv::Mat disparity(rows, cols, CV_8UC1, h_disparity);
	return disparity;
}

static void free_memory() 
{
	CUDA_CHECK_RETURN(hipFree(d_im0));
	CUDA_CHECK_RETURN(hipFree(d_im1));
	CUDA_CHECK_RETURN(hipFree(d_transform0));
	CUDA_CHECK_RETURN(hipFree(d_transform1));
	CUDA_CHECK_RETURN(hipFree(d_L0));
	CUDA_CHECK_RETURN(hipFree(d_L1));
	CUDA_CHECK_RETURN(hipFree(d_L2));
	CUDA_CHECK_RETURN(hipFree(d_L3));
	CUDA_CHECK_RETURN(hipFree(d_disparity));
	CUDA_CHECK_RETURN(hipFree(d_disparity_filtered_uchar));
	CUDA_CHECK_RETURN(hipFree(d_cost));

	delete[] h_disparity;
}

void finish_disparity_method(bool &first_alloc) 
{
	if(!first_alloc) 
	{
		free_memory();
		CUDA_CHECK_RETURN(hipStreamDestroy(stream1));
		CUDA_CHECK_RETURN(hipStreamDestroy(stream2));
		CUDA_CHECK_RETURN(hipStreamDestroy(stream3));
	}
}
